#include "hip/hip_runtime.h"
#include "utile.h"

#include "camera.h"
#include "hittable.h"
#include "hittable_list.h"
#include "sphere.h"

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__constant__ int d_samples_per_pixel;
__constant__ float d_pixel_samples_scale;

// Initialize cuRAND state for a single thread
__global__ void rand_init(hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);            // Custom seed : 1984
    }
}

// Initialize cuRAND state for each pixel, ensuring a unique random sequence for each pixel
__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;      // range [0, 1200]
    int j = threadIdx.y + blockIdx.y * blockDim.y;      // range [0, 679]

    if ((i >= max_x) || (j >= max_y)) return;

    int pixel_index = j * max_x + i;
    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);    // Each thread gets different seed number (Each ramdom number generation pattern must be independent per each thread)
}

__global__ void render(vec3* fb, int max_x, int max_y, int samples_per_pixel, camera** cam, hittable ** d_world, hiprandState* rand_state) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int depth = 50;
    int pixel_index = row * max_x + col;
    hiprandState local_rand_state = rand_state[pixel_index];

    if ((col >= max_x) || (row >= max_y)) return;

    float u = float(col) / float(max_x);
    float v = float(row) / float(max_y);

    color pixel_color(0, 0, 0); 
    for (int sample = 0; sample < samples_per_pixel; sample++) {
        ray r = (*cam)->get_ray(col, row, &local_rand_state);
        pixel_color += ray_color(r, d_world, &local_rand_state);
    }
    pixel_color /= float(samples_per_pixel);
    
    pixel_color[0] = linear_to_gamma(pixel_color[0]);
    pixel_color[1] = linear_to_gamma(pixel_color[1]);
    pixel_color[2] = linear_to_gamma(pixel_color[2]);
        
    fb[pixel_index] = pixel_color;
}

//__global__ void render(vec3* fb, int max_x, int max_y, camera** cam, hittable** d_world) {
//    // 1D ������� ��� �������� �۷ι� �ε��� ���
//    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
//
//    if (thread_id >= max_x * max_y) return;
//
//    // 1D �ε����� 2D (col, row)�� ��ȯw 1
//    int col = thread_id % max_x;  // �� = thread_id�� �̹��� ���� ũ��� ���� ������
//    int row = thread_id / max_x;  // �� = thread_id�� �̹��� ���� ũ��� ���� ��
//
//    // �ȼ� �ε��� ��� (1D �迭 ���ٿ�)
//    int pixel_index = row * max_x + col;
//
//    // u�� v ���
//    float u = float(col) / float(max_x);
//    float v = float(row) / float(max_y);
//
//    // Ray ���� �� ���� ���
//    ray r = (*cam)->get_ray(col, row);
//    fb[pixel_index] = ray_color(r, d_world);
//}

__global__ void create_world(hittable** objects, hittable** d_world, camera** cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        objects[0] = new sphere(vec3(0, 0, -1), 0.5);
        objects[1] = new sphere(vec3(0, -100.5, -1), 100);
        *d_world = new hittable_list(objects, 2);
        *cam = new camera();
    }
}

__global__ void free_world(hittable** d_object_list, hittable** d_world, camera** cam) {
    delete d_object_list[0];
    delete d_object_list[1];
    delete* d_world;
    delete* cam;
}

int main() {
    auto aspect_ratio = 16.0 / 9.0;
    int image_width = 1200;
    int tx = 8;                                  
    int ty = 8;
    int samples_per_pixel = 500;

    // Calculate the image height, and ensure that it's at least 1.
    int image_height = int(image_width / aspect_ratio);
    image_height = (image_height < 1) ? 1 : image_height;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "with one thread per pixel.\n";

    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(vec3);

    // Allocate random states
    hiprandState* d_rand_state_1;                // For rendering        
    checkCudaErrors(hipMalloc((void**)&d_rand_state_1, num_pixels * sizeof(hiprandState)));
    hiprandState* d_rand_state_2;                // For world creation
    checkCudaErrors(hipMalloc((void**)&d_rand_state_2, 1 * sizeof(hiprandState)));

    // Allocate memory on CPU and GPU
    vec3* d_fb;                        // Device memory
    checkCudaErrors(hipMalloc((void**)&d_fb, fb_size));

    rand_init << <1, 1 >> > (d_rand_state_2);   // 2nd random state initialization
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    hittable** d_object_list;
    int num_hittables = 3;         // Total object number
    checkCudaErrors(hipMalloc((void**)&d_object_list, num_hittables * sizeof(hittable*)));

    hittable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*)));

    camera** cam;
    checkCudaErrors(hipMalloc((void**)&cam, sizeof(camera*)));

    // make our world of hittables
    create_world << <1, 1 >> > (d_object_list, d_world, cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // CUDA Events for timing
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // Start recording time
    checkCudaErrors(hipEventRecord(start, 0));

    // Render our buffer
    // �� �Ǵ°�
    //int threads_per_block = 256; // Optimal number of threads per block
    //int blocks_per_grid = (num_pixels + threads_per_block - 1) / threads_per_block; // Calculate grid size
    //render << <blocks_per_grid, threads_per_block >> > (d_fb, image_width, image_height, cam, d_world);

    // �Ϲ���c

    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);

    render_init << <blocks, threads >> > (image_width, image_height, d_rand_state_1);   // 1st random state initialization
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render << <blocks, threads >> > (d_fb, image_width, image_height, samples_per_pixel, cam, d_world, d_rand_state_1);

    // �� �Ǵ°� ����
    //int threads_per_block = 256;
    //int blocks_per_grid = (image_width * image_height + threads_per_block - 1) / threads_per_block;
    //render << <blocks_per_grid, threads_per_block >> > (d_fb, image_width, image_height, cam, d_world);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Copy results back to host
    vec3* h_fb = (vec3*)malloc(fb_size); // Host memory
    checkCudaErrors(hipMemcpy(h_fb, d_fb, fb_size, hipMemcpyDeviceToHost));


    /////////////////////////////////////////////////////////////////////////////
    // Output File
    /////////////////////////////////////////////////////////////////////////////

    FILE* f = fopen("image3.ppm", "w");
    std::fprintf(f, "P3\n%d %d\n%d\n", image_width, image_height, 255);
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            auto ir = h_fb[pixel_index].r();
            auto ig = h_fb[pixel_index].g();
            auto ib = h_fb[pixel_index].b();


            static const interval intensity(0.000, 0.999);
            int rbyte = int(256 * intensity.clamp(ir));
            int gbyte = int(256 * intensity.clamp(ig));
            int bbyte = int(256 * intensity.clamp(ib));
            std::fprintf(f, "%d %d %d ", rbyte, gbyte, bbyte);
        }
    }
    std::clog << "\rDone.                 \n";

    /////////////////////////////////////////////////////////////////////////////
    // Output Console
    /////////////////////////////////////////////////////////////////////////////

    //std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    //for (int j = image_height - 1; j >= 0; j--) {
    //    for (int i = 0; i < image_width; i++) {
    //        size_t pixel_index = j * image_width + i;
    //        int ir = int(255.99 * h_fb[pixel_index].r());
    //        int ig = int(255.99 * h_fb[pixel_index].g());
    //        int ib = int(255.99 * h_fb[pixel_index].b());
    //        std::cout << ir << " " << ig << " " << ib << "\n";
    //    }
    //}

    //std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    //for (int j = image_height - 1; j >= 0; j--) {
    //    for (int i = 0; i < image_width; i++) {
    //        size_t pixel_index = j * image_width + i;
    //        auto ir = h_fb[pixel_index].r();
    //        auto ig = h_fb[pixel_index].g();
    //        auto ib = h_fb[pixel_index].b();

    //        static const interval intensity(0.000, 0.999);
    //        int rbyte = int(255.99 * intensity.clamp(ir));
    //        int gbyte = int(255.99 * intensity.clamp(ig));
    //        int bbyte = int(255.99 * intensity.clamp(ib));
    //        std::cout << rbyte << " " << gbyte << " " << bbyte << "\n";
    //    }
    //}

    // Stop recording time
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    // Calculate elapsed time
    float milliseconds = 0;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));
    std::cerr << "Rendering took " << milliseconds / 1000.0f << " seconds.\n";

    // Destroy CUDA Events
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (d_object_list, d_world, cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_object_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_fb));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}
